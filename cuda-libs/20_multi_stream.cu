// Multi-Stream Concurrent Execution: Maximize GPU utilization
// Critical for overlapping computation, memory transfers, and pipelining

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s (line %d)\n", hipGetErrorString(err), __LINE__); \
        exit(EXIT_FAILURE); \
    } \
}

// Simple kernel for demonstration
__global__ void vectorAdd(float* a, float* b, float* c, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}

// Computation-intensive kernel
__global__ void matrixCompute(float* data, int n, int iterations) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        float value = data[tid];
        for (int i = 0; i < iterations; i++) {
            value = sinf(value) * cosf(value) + value * 0.5f;
        }
        data[tid] = value;
    }
}

int main() {
    const int NUM_STREAMS = 4;
    const int N = 1 << 24;  // 16M elements per stream
    const int bytes_per_stream = N * sizeof(float);

    printf("=== Multi-Stream Concurrent Execution ===\n");
    printf("Number of streams: %d\n", NUM_STREAMS);
    printf("Elements per stream: %d\n", N);
    printf("Memory per stream: %.2f MB\n\n", bytes_per_stream / 1024.0 / 1024.0);

    // Create streams
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        CHECK_CUDA(hipStreamCreate(&streams[i]));
    }

    // Allocate pinned host memory for faster transfers
    float *h_a[NUM_STREAMS], *h_b[NUM_STREAMS], *h_c[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        CHECK_CUDA(hipHostMalloc(&h_a[i], bytes_per_stream, hipHostMallocDefault));
        CHECK_CUDA(hipHostMalloc(&h_b[i], bytes_per_stream, hipHostMallocDefault));
        CHECK_CUDA(hipHostMalloc(&h_c[i], bytes_per_stream, hipHostMallocDefault));

        // Initialize data
        for (int j = 0; j < N; j++) {
            h_a[i][j] = (float)(rand() % 100);
            h_b[i][j] = (float)(rand() % 100);
        }
    }

    // Allocate device memory
    float *d_a[NUM_STREAMS], *d_b[NUM_STREAMS], *d_c[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        CHECK_CUDA(hipMalloc(&d_a[i], bytes_per_stream));
        CHECK_CUDA(hipMalloc(&d_b[i], bytes_per_stream));
        CHECK_CUDA(hipMalloc(&d_c[i], bytes_per_stream));
    }

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // ===== SEQUENTIAL EXECUTION (Baseline) =====
    printf("=== Sequential Execution (No Streams) ===\n");

    CHECK_CUDA(hipEventRecord(start));

    for (int i = 0; i < NUM_STREAMS; i++) {
        CHECK_CUDA(hipMemcpy(d_a[i], h_a[i], bytes_per_stream, hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(d_b[i], h_b[i], bytes_per_stream, hipMemcpyHostToDevice));
        vectorAdd<<<gridSize, blockSize>>>(d_a[i], d_b[i], d_c[i], N);
        CHECK_CUDA(hipMemcpy(h_c[i], d_c[i], bytes_per_stream, hipMemcpyDeviceToHost));
    }

    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float ms_sequential = 0;
    CHECK_CUDA(hipEventElapsedTime(&ms_sequential, start, stop));

    printf("Total time: %.3f ms\n\n", ms_sequential);

    // ===== CONCURRENT EXECUTION WITH STREAMS =====
    printf("=== Concurrent Execution (Multi-Stream) ===\n");

    CHECK_CUDA(hipEventRecord(start));

    // Launch all operations asynchronously
    for (int i = 0; i < NUM_STREAMS; i++) {
        CHECK_CUDA(hipMemcpyAsync(d_a[i], h_a[i], bytes_per_stream,
                                   hipMemcpyHostToDevice, streams[i]));
        CHECK_CUDA(hipMemcpyAsync(d_b[i], h_b[i], bytes_per_stream,
                                   hipMemcpyHostToDevice, streams[i]));

        vectorAdd<<<gridSize, blockSize, 0, streams[i]>>>(d_a[i], d_b[i], d_c[i], N);

        CHECK_CUDA(hipMemcpyAsync(h_c[i], d_c[i], bytes_per_stream,
                                   hipMemcpyDeviceToHost, streams[i]));
    }

    // Wait for all streams to complete
    for (int i = 0; i < NUM_STREAMS; i++) {
        CHECK_CUDA(hipStreamSynchronize(streams[i]));
    }

    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float ms_concurrent = 0;
    CHECK_CUDA(hipEventElapsedTime(&ms_concurrent, start, stop));

    printf("Total time: %.3f ms\n", ms_concurrent);
    printf("Speedup: %.2fx\n\n", ms_sequential / ms_concurrent);

    // ===== PIPELINED EXECUTION =====
    printf("=== Pipelined Execution (Staged) ===\n");

    const int NUM_CHUNKS = 8;
    const int chunk_size = N / NUM_CHUNKS;
    const int chunk_bytes = chunk_size * sizeof(float);

    CHECK_CUDA(hipEventRecord(start));

    for (int chunk = 0; chunk < NUM_CHUNKS; chunk++) {
        int stream_id = chunk % NUM_STREAMS;
        int offset = chunk * chunk_size;

        CHECK_CUDA(hipMemcpyAsync(d_a[stream_id] + offset, h_a[stream_id] + offset,
                                   chunk_bytes, hipMemcpyHostToDevice, streams[stream_id]));
        CHECK_CUDA(hipMemcpyAsync(d_b[stream_id] + offset, h_b[stream_id] + offset,
                                   chunk_bytes, hipMemcpyHostToDevice, streams[stream_id]));

        int chunk_grid = (chunk_size + blockSize - 1) / blockSize;
        vectorAdd<<<chunk_grid, blockSize, 0, streams[stream_id]>>>(
            d_a[stream_id] + offset, d_b[stream_id] + offset, d_c[stream_id] + offset, chunk_size);

        CHECK_CUDA(hipMemcpyAsync(h_c[stream_id] + offset, d_c[stream_id] + offset,
                                   chunk_bytes, hipMemcpyDeviceToHost, streams[stream_id]));
    }

    for (int i = 0; i < NUM_STREAMS; i++) {
        CHECK_CUDA(hipStreamSynchronize(streams[i]));
    }

    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float ms_pipelined = 0;
    CHECK_CUDA(hipEventElapsedTime(&ms_pipelined, start, stop));

    printf("Total time: %.3f ms\n", ms_pipelined);
    printf("Speedup: %.2fx\n\n", ms_sequential / ms_pipelined);

    // ===== STREAM PRIORITIES =====
    printf("=== Stream Priorities ===\n");

    int priority_low, priority_high;
    CHECK_CUDA(hipDeviceGetStreamPriorityRange(&priority_low, &priority_high));

    hipStream_t high_priority_stream, low_priority_stream;
    CHECK_CUDA(hipStreamCreateWithPriority(&high_priority_stream, hipStreamNonBlocking, priority_high));
    CHECK_CUDA(hipStreamCreateWithPriority(&low_priority_stream, hipStreamNonBlocking, priority_low));

    printf("Priority range: %d (high) to %d (low)\n", priority_high, priority_low);

    // Launch computation-heavy kernel on low priority
    matrixCompute<<<gridSize, blockSize, 0, low_priority_stream>>>(d_a[0], N, 100);

    // Launch quick kernel on high priority
    vectorAdd<<<gridSize, blockSize, 0, high_priority_stream>>>(d_a[1], d_b[1], d_c[1], N);

    CHECK_CUDA(hipStreamSynchronize(high_priority_stream));
    CHECK_CUDA(hipStreamSynchronize(low_priority_stream));

    printf("Stream priorities demonstrated successfully!\n\n");

    // ===== STREAM DEPENDENCIES (EVENTS) =====
    printf("=== Stream Dependencies with Events ===\n");

    hipEvent_t event1, event2;
    CHECK_CUDA(hipEventCreate(&event1));
    CHECK_CUDA(hipEventCreate(&event2));

    CHECK_CUDA(hipEventRecord(start));

    // Stream 0: First operation
    vectorAdd<<<gridSize, blockSize, 0, streams[0]>>>(d_a[0], d_b[0], d_c[0], N);
    CHECK_CUDA(hipEventRecord(event1, streams[0]));

    // Stream 1: Wait for stream 0 to complete
    CHECK_CUDA(hipStreamWaitEvent(streams[1], event1, 0));
    vectorAdd<<<gridSize, blockSize, 0, streams[1]>>>(d_c[0], d_b[1], d_c[1], N);
    CHECK_CUDA(hipEventRecord(event2, streams[1]));

    // Stream 2: Wait for stream 1 to complete
    CHECK_CUDA(hipStreamWaitEvent(streams[2], event2, 0));
    vectorAdd<<<gridSize, blockSize, 0, streams[2]>>>(d_c[1], d_b[2], d_c[2], N);

    CHECK_CUDA(hipStreamSynchronize(streams[2]));

    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float ms_dependent = 0;
    CHECK_CUDA(hipEventElapsedTime(&ms_dependent, start, stop));

    printf("Dependent stream execution: %.3f ms\n\n", ms_dependent);

    // Performance summary
    printf("=== Performance Summary ===\n");
    printf("Sequential:   %.3f ms\n", ms_sequential);
    printf("Concurrent:   %.3f ms (%.2fx speedup)\n", ms_concurrent, ms_sequential / ms_concurrent);
    printf("Pipelined:    %.3f ms (%.2fx speedup)\n", ms_pipelined, ms_sequential / ms_pipelined);
    printf("Dependent:    %.3f ms\n", ms_dependent);

    printf("\nTotal data processed: %.2f GB\n",
           (NUM_STREAMS * N * 3 * sizeof(float)) / 1024.0 / 1024.0 / 1024.0);

    // Cleanup
    for (int i = 0; i < NUM_STREAMS; i++) {
        CHECK_CUDA(hipStreamDestroy(streams[i]));
        CHECK_CUDA(hipHostFree(h_a[i]));
        CHECK_CUDA(hipHostFree(h_b[i]));
        CHECK_CUDA(hipHostFree(h_c[i]));
        CHECK_CUDA(hipFree(d_a[i]));
        CHECK_CUDA(hipFree(d_b[i]));
        CHECK_CUDA(hipFree(d_c[i]));
    }

    CHECK_CUDA(hipStreamDestroy(high_priority_stream));
    CHECK_CUDA(hipStreamDestroy(low_priority_stream));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    CHECK_CUDA(hipEventDestroy(event1));
    CHECK_CUDA(hipEventDestroy(event2));

    printf("\nMulti-stream operations completed successfully!\n");
    return 0;
}
