// cuSPARSE SpMM: Sparse Matrix-Matrix multiplication
// Used in graph convolutions, attention mechanisms, and large-scale ML models

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s (line %d)\n", hipGetErrorString(err), __LINE__); \
        exit(EXIT_FAILURE); \
    } \
}

#define CHECK_CUSPARSE(call) { \
    hipsparseStatus_t status = call; \
    if (status != HIPSPARSE_STATUS_SUCCESS) { \
        fprintf(stderr, "cuSPARSE Error: %d (line %d)\n", status, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
}

int main() {
    // Matrix dimensions: C(M x N) = A(M x K) * B(K x N)
    const int M = 4096;
    const int K = 4096;
    const int N = 128;  // Multiple right-hand sides (as in batch operations)
    const float density = 0.05f;  // 5% sparse

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Generate sparse matrix A in CSR format
    int nnz = (int)(M * K * density);
    int *h_csrRowPtr = (int*)malloc((M + 1) * sizeof(int));
    int *h_csrColIdx = (int*)malloc(nnz * sizeof(int));
    float *h_csrValues = (float*)malloc(nnz * sizeof(float));

    // Simple sparse pattern generation
    int nnz_per_row = nnz / M;
    h_csrRowPtr[0] = 0;
    for (int i = 0; i < M; i++) {
        h_csrRowPtr[i + 1] = h_csrRowPtr[i] + nnz_per_row;
    }
    h_csrRowPtr[M] = nnz;  // Adjust last element

    for (int i = 0; i < nnz; i++) {
        h_csrColIdx[i] = rand() % K;
        h_csrValues[i] = (float)(rand() % 100) / 10.0f;
    }

    // Generate dense matrix B
    float *h_B = (float*)malloc(K * N * sizeof(float));
    float *h_C = (float*)malloc(M * N * sizeof(float));

    for (int i = 0; i < K * N; i++) {
        h_B[i] = (float)(rand() % 100) / 10.0f;
    }

    // Device memory
    int *d_csrRowPtr, *d_csrColIdx;
    float *d_csrValues, *d_B, *d_C;

    CHECK_CUDA(hipMalloc(&d_csrRowPtr, (M + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_csrColIdx, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_csrValues, nnz * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B, K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_C, M * N * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_csrRowPtr, h_csrRowPtr, (M + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_csrColIdx, h_csrColIdx, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_csrValues, h_csrValues, nnz * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice));

    // Create cuSPARSE handle
    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Create sparse matrix A
    hipsparseSpMatDescr_t matA;
    CHECK_CUSPARSE(hipsparseCreateCsr(&matA, M, K, nnz,
                                     d_csrRowPtr, d_csrColIdx, d_csrValues,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    // Create dense matrices B and C
    hipsparseDnMatDescr_t matB, matC;
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, K, N, N, d_B, HIP_R_32F, HIPSPARSE_ORDER_ROW));
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matC, M, N, N, d_C, HIP_R_32F, HIPSPARSE_ORDER_ROW));

    // Allocate buffer
    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha, matA, matB, &beta, matC,
                                           HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT,
                                           &bufferSize));

    void* buffer = NULL;
    CHECK_CUDA(hipMalloc(&buffer, bufferSize));

    // Warm-up
    CHECK_CUSPARSE(hipsparseSpMM(handle,
                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha, matA, matB, &beta, matC,
                               HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT,
                               buffer));

    // Timing
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    CHECK_CUDA(hipEventRecord(start));

    // SpMM: C = alpha * A * B + beta * C
    CHECK_CUSPARSE(hipsparseSpMM(handle,
                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha, matA, matB, &beta, matC,
                               HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT,
                               buffer));

    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));

    // Copy result
    CHECK_CUDA(hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // Performance metrics
    double gflops = (2.0 * nnz * N * 1e-9) / (milliseconds / 1000.0);

    printf("=== Sparse Matrix-Matrix Multiplication ===\n");
    printf("C(%dx%d) = A(%dx%d, %.1f%% sparse) * B(%dx%d)\n",
           M, N, M, K, density * 100, K, N);
    printf("Non-zeros in A: %d\n", nnz);
    printf("Time: %.3f ms\n", milliseconds);
    printf("Performance: %.2f GFLOPS\n", gflops);

    // Display sample result
    printf("\nResult C (4x4 corner):\n");
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            printf("%.2f ", h_C[i * N + j]);
        }
        printf("\n");
    }

    // Cleanup
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matB));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matC));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    CHECK_CUDA(hipFree(d_csrRowPtr));
    CHECK_CUDA(hipFree(d_csrColIdx));
    CHECK_CUDA(hipFree(d_csrValues));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipFree(buffer));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    free(h_csrRowPtr);
    free(h_csrColIdx);
    free(h_csrValues);
    free(h_B);
    free(h_C);

    printf("\ncuSPARSE SpMM completed successfully!\n");
    return 0;
}
