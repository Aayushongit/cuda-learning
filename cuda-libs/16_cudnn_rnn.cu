// cuDNN RNN/LSTM: Recurrent neural networks for sequence processing
// Essential for NLP, time-series, and sequential data processing

#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s (line %d)\n", hipGetErrorString(err), __LINE__); \
        exit(EXIT_FAILURE); \
    } \
}

#define CHECK_CUDNN(call) { \
    hipdnnStatus_t status = call; \
    if (status != HIPDNN_STATUS_SUCCESS) { \
        fprintf(stderr, "cuDNN Error: %s (line %d)\n", hipdnnGetErrorString(status), __LINE__); \
        exit(EXIT_FAILURE); \
    } \
}

int main() {
    // RNN configuration
    const int seq_length = 32;      // Sequence length
    const int batch_size = 64;      // Batch size
    const int input_size = 128;     // Input feature dimension
    const int hidden_size = 256;    // Hidden state size
    const int num_layers = 2;       // Number of stacked RNN layers

    printf("=== cuDNN LSTM Network ===\n");
    printf("Sequence length: %d\n", seq_length);
    printf("Batch size:      %d\n", batch_size);
    printf("Input size:      %d\n", input_size);
    printf("Hidden size:     %d\n", hidden_size);
    printf("Num layers:      %d\n\n", num_layers);

    // Create cuDNN handle
    hipdnnHandle_t cudnn;
    CHECK_CUDNN(hipdnnCreate(&cudnn));

    // Create RNN descriptor
    hipdnnRNNDescriptor_t rnn_desc;
    hipdnnDropoutDescriptor_t dropout_desc;
    CHECK_CUDNN(hipdnnCreateRNNDescriptor(&rnn_desc));
    CHECK_CUDNN(hipdnnCreateDropoutDescriptor(&dropout_desc));

    // Setup dropout (disabled for simplicity)
    size_t state_size;
    CHECK_CUDNN(hipdnnDropoutGetStatesSize(cudnn, &state_size));
    void *dropout_states;
    CHECK_CUDA(hipMalloc(&dropout_states, state_size));

    CHECK_CUDNN(hipdnnSetDropoutDescriptor(dropout_desc, cudnn,
                                          0.0f,  // dropout probability (0 = disabled)
                                          dropout_states, state_size,
                                          1234ULL));  // seed

    // Set RNN descriptor for LSTM
    CHECK_CUDNN(hipdnnSetRNNDescriptor_v6(cudnn, rnn_desc,
                                         hidden_size,
                                         num_layers,
                                         dropout_desc,
                                         HIPDNN_LINEAR_INPUT,
                                         HIPDNN_UNIDIRECTIONAL,
                                         HIPDNN_LSTM,
                                         HIPDNN_RNN_ALGO_STANDARD,
                                         HIPDNN_DATA_FLOAT));

    // Create tensor descriptors for each time step
    hipdnnTensorDescriptor_t *x_desc = (hipdnnTensorDescriptor_t*)malloc(seq_length * sizeof(hipdnnTensorDescriptor_t));
    hipdnnTensorDescriptor_t *y_desc = (hipdnnTensorDescriptor_t*)malloc(seq_length * sizeof(hipdnnTensorDescriptor_t));

    int dims_x[3] = {batch_size, input_size, 1};
    int strides_x[3] = {dims_x[1] * dims_x[2], dims_x[2], 1};

    int dims_y[3] = {batch_size, hidden_size, 1};
    int strides_y[3] = {dims_y[1] * dims_y[2], dims_y[2], 1};

    for (int i = 0; i < seq_length; i++) {
        CHECK_CUDNN(hipdnnCreateTensorDescriptor(&x_desc[i]));
        CHECK_CUDNN(hipdnnSetTensorNdDescriptor(x_desc[i], HIPDNN_DATA_FLOAT, 3, dims_x, strides_x));

        CHECK_CUDNN(hipdnnCreateTensorDescriptor(&y_desc[i]));
        CHECK_CUDNN(hipdnnSetTensorNdDescriptor(y_desc[i], HIPDNN_DATA_FLOAT, 3, dims_y, strides_y));
    }

    // Hidden and cell state descriptors
    hipdnnTensorDescriptor_t h_desc, c_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&h_desc));
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&c_desc));

    int dims_hc[3] = {num_layers, batch_size, hidden_size};
    int strides_hc[3] = {dims_hc[1] * dims_hc[2], dims_hc[2], 1};

    CHECK_CUDNN(hipdnnSetTensorNdDescriptor(h_desc, HIPDNN_DATA_FLOAT, 3, dims_hc, strides_hc));
    CHECK_CUDNN(hipdnnSetTensorNdDescriptor(c_desc, HIPDNN_DATA_FLOAT, 3, dims_hc, strides_hc));

    // Get weight space size
    size_t weight_space_size;
    CHECK_CUDNN(hipdnnGetRNNParamsSize(cudnn, rnn_desc, x_desc[0], &weight_space_size, HIPDNN_DATA_FLOAT));

    printf("Weight space size: %.2f MB\n\n", weight_space_size / 1024.0 / 1024.0);

    // Allocate memory
    size_t input_bytes = seq_length * batch_size * input_size * sizeof(float);
    size_t output_bytes = seq_length * batch_size * hidden_size * sizeof(float);
    size_t hidden_bytes = num_layers * batch_size * hidden_size * sizeof(float);

    float *h_input = (float*)malloc(input_bytes);
    float *h_output = (float*)malloc(output_bytes);

    // Initialize input with random data
    for (size_t i = 0; i < seq_length * batch_size * input_size; i++) {
        h_input[i] = (float)(rand() % 100) / 100.0f;
    }

    float *d_input, *d_output;
    float *d_hx, *d_cx, *d_hy, *d_cy;
    void *d_weights;

    CHECK_CUDA(hipMalloc(&d_input, input_bytes));
    CHECK_CUDA(hipMalloc(&d_output, output_bytes));
    CHECK_CUDA(hipMalloc(&d_hx, hidden_bytes));
    CHECK_CUDA(hipMalloc(&d_cx, hidden_bytes));
    CHECK_CUDA(hipMalloc(&d_hy, hidden_bytes));
    CHECK_CUDA(hipMalloc(&d_cy, hidden_bytes));
    CHECK_CUDA(hipMalloc(&d_weights, weight_space_size));

    CHECK_CUDA(hipMemcpy(d_input, h_input, input_bytes, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemset(d_hx, 0, hidden_bytes));
    CHECK_CUDA(hipMemset(d_cx, 0, hidden_bytes));
    CHECK_CUDA(hipMemset(d_weights, 0, weight_space_size));

    // Get workspace size
    size_t workspace_size;
    CHECK_CUDNN(hipdnnGetRNNWorkspaceSize(cudnn, rnn_desc, seq_length, x_desc, &workspace_size));

    void *d_workspace;
    CHECK_CUDA(hipMalloc(&d_workspace, workspace_size));

    printf("Workspace size: %.2f MB\n\n", workspace_size / 1024.0 / 1024.0);

    // Timing
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Create array of input/output pointers
    float **d_x = (float**)malloc(seq_length * sizeof(float*));
    float **d_y = (float**)malloc(seq_length * sizeof(float*));

    for (int i = 0; i < seq_length; i++) {
        d_x[i] = d_input + i * batch_size * input_size;
        d_y[i] = d_output + i * batch_size * hidden_size;
    }

    // Warm-up
    CHECK_CUDNN(hipdnnRNNForwardInference(cudnn, rnn_desc,
                                         seq_length,
                                         x_desc, d_input,
                                         h_desc, d_hx,
                                         c_desc, d_cx,
                                         rnn_desc, d_weights,
                                         y_desc, d_output,
                                         h_desc, d_hy,
                                         c_desc, d_cy,
                                         d_workspace, workspace_size));

    // Forward pass
    CHECK_CUDA(hipEventRecord(start));

    CHECK_CUDNN(hipdnnRNNForwardInference(cudnn, rnn_desc,
                                         seq_length,
                                         x_desc, d_input,
                                         h_desc, d_hx,
                                         c_desc, d_cx,
                                         rnn_desc, d_weights,
                                         y_desc, d_output,
                                         h_desc, d_hy,
                                         c_desc, d_cy,
                                         d_workspace, workspace_size));

    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));

    // Copy results
    CHECK_CUDA(hipMemcpy(h_output, d_output, output_bytes, hipMemcpyDeviceToHost));

    // Performance metrics
    printf("=== Performance ===\n");
    printf("Forward pass time: %.3f ms\n", milliseconds);
    printf("Sequences/sec:     %.2f\n", batch_size / (milliseconds / 1000.0));
    printf("Throughput:        %.2f M elements/sec\n",
           (seq_length * batch_size * hidden_size) / (milliseconds * 1000.0));

    // Sample output
    printf("\nOutput sample (first timestep, first 10 hidden units):\n");
    for (int i = 0; i < 10; i++) {
        printf("%.4f ", h_output[i]);
    }

    printf("\n\nOutput sample (last timestep, first 10 hidden units):\n");
    int last_timestep = (seq_length - 1) * batch_size * hidden_size;
    for (int i = 0; i < 10; i++) {
        printf("%.4f ", h_output[last_timestep + i]);
    }
    printf("\n");

    // Cleanup
    for (int i = 0; i < seq_length; i++) {
        CHECK_CUDNN(hipdnnDestroyTensorDescriptor(x_desc[i]));
        CHECK_CUDNN(hipdnnDestroyTensorDescriptor(y_desc[i]));
    }
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(h_desc));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(c_desc));
    CHECK_CUDNN(hipdnnDestroyRNNDescriptor(rnn_desc));
    CHECK_CUDNN(hipdnnDestroyDropoutDescriptor(dropout_desc));
    CHECK_CUDNN(hipdnnDestroy(cudnn));

    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_output));
    CHECK_CUDA(hipFree(d_hx));
    CHECK_CUDA(hipFree(d_cx));
    CHECK_CUDA(hipFree(d_hy));
    CHECK_CUDA(hipFree(d_cy));
    CHECK_CUDA(hipFree(d_weights));
    CHECK_CUDA(hipFree(d_workspace));
    CHECK_CUDA(hipFree(dropout_states));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    free(x_desc);
    free(y_desc);
    free(d_x);
    free(d_y);
    free(h_input);
    free(h_output);

    printf("\ncuDNN RNN/LSTM completed successfully!\n");
    return 0;
}
