#include "hip/hip_runtime.h"
// CUDA Graphs: Reduce kernel launch overhead by recording execution patterns
// Can reduce CPU overhead from ~10μs to <1μs per kernel launch

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s (line %d)\n", hipGetErrorString(err), __LINE__); \
        exit(EXIT_FAILURE); \
    } \
}

__global__ void vectorAdd(float* a, float* b, float* c, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}

__global__ void vectorMul(float* a, float* b, float* c, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        c[tid] = a[tid] * b[tid];
    }
}

__global__ void vectorScale(float* a, float scale, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        a[tid] *= scale;
    }
}

int main() {
    const int N = 1 << 20;  // 1M elements
    const int iterations = 1000;
    const size_t bytes = N * sizeof(float);

    printf("=== CUDA Graphs Optimization ===\n");
    printf("Array size: %d elements\n", N);
    printf("Iterations: %d\n\n", iterations);

    // Allocate memory
    float *d_a, *d_b, *d_c, *d_temp;
    CHECK_CUDA(hipMalloc(&d_a, bytes));
    CHECK_CUDA(hipMalloc(&d_b, bytes));
    CHECK_CUDA(hipMalloc(&d_c, bytes));
    CHECK_CUDA(hipMalloc(&d_temp, bytes));

    // Initialize
    CHECK_CUDA(hipMemset(d_a, 1, bytes));
    CHECK_CUDA(hipMemset(d_b, 2, bytes));

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // ===== Method 1: Traditional kernel launches =====
    printf("=== Traditional Kernel Launches ===\n");

    CHECK_CUDA(hipEventRecord(start));

    for (int i = 0; i < iterations; i++) {
        vectorAdd<<<gridSize, blockSize>>>(d_a, d_b, d_temp, N);
        vectorMul<<<gridSize, blockSize>>>(d_temp, d_b, d_c, N);
        vectorScale<<<gridSize, blockSize>>>(d_c, 0.5f, N);
    }

    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipEventSynchronize(stop));

    float ms_traditional = 0;
    CHECK_CUDA(hipEventElapsedTime(&ms_traditional, start, stop));

    printf("Total time: %.3f ms\n", ms_traditional);
    printf("Avg per iteration: %.3f ms\n", ms_traditional / iterations);
    printf("Launch overhead: ~%.2f μs per kernel\n\n",
           (ms_traditional / iterations / 3.0) * 1000.0);

    // ===== Method 2: CUDA Graphs with Stream Capture =====
    printf("=== CUDA Graphs (Stream Capture) ===\n");

    hipGraph_t graph;
    hipGraphExec_t graphExec;
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));

    // Capture the sequence of operations
    CHECK_CUDA(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));

    vectorAdd<<<gridSize, blockSize, 0, stream>>>(d_a, d_b, d_temp, N);
    vectorMul<<<gridSize, blockSize, 0, stream>>>(d_temp, d_b, d_c, N);
    vectorScale<<<gridSize, blockSize, 0, stream>>>(d_c, 0.5f, N);

    CHECK_CUDA(hipStreamEndCapture(stream, &graph));

    // Instantiate the graph
    CHECK_CUDA(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

    // Benchmark graph execution
    CHECK_CUDA(hipEventRecord(start));

    for (int i = 0; i < iterations; i++) {
        CHECK_CUDA(hipGraphLaunch(graphExec, stream));
    }

    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipStreamSynchronize(stream));
    CHECK_CUDA(hipEventSynchronize(stop));

    float ms_graph = 0;
    CHECK_CUDA(hipEventElapsedTime(&ms_graph, start, stop));

    printf("Total time: %.3f ms\n", ms_graph);
    printf("Avg per iteration: %.3f ms\n", ms_graph / iterations);
    printf("Launch overhead: ~%.2f μs per graph\n", (ms_graph / iterations) * 1000.0);
    printf("Speedup: %.2fx\n\n", ms_traditional / ms_graph);

    // ===== Method 3: Manual Graph Construction =====
    printf("=== CUDA Graphs (Manual Construction) ===\n");

    hipGraph_t manual_graph;
    hipGraphExec_t manual_graphExec;
    CHECK_CUDA(hipGraphCreate(&manual_graph, 0));

    // Create nodes manually
    hipGraphNode_t addNode, mulNode, scaleNode;
    hipKernelNodeParams addParams = {0};
    hipKernelNodeParams mulParams = {0};
    hipKernelNodeParams scaleParams = {0};

    // Configure add kernel
    void* addArgs[] = {&d_a, &d_b, &d_temp, &N};
    addParams.func = (void*)vectorAdd;
    addParams.gridDim = dim3(gridSize);
    addParams.blockDim = dim3(blockSize);
    addParams.kernelParams = addArgs;

    // Configure mul kernel
    void* mulArgs[] = {&d_temp, &d_b, &d_c, &N};
    mulParams.func = (void*)vectorMul;
    mulParams.gridDim = dim3(gridSize);
    mulParams.blockDim = dim3(blockSize);
    mulParams.kernelParams = mulArgs;

    // Configure scale kernel
    float scale = 0.5f;
    void* scaleArgs[] = {&d_c, &scale, &N};
    scaleParams.func = (void*)vectorScale;
    scaleParams.gridDim = dim3(gridSize);
    scaleParams.blockDim = dim3(blockSize);
    scaleParams.kernelParams = scaleArgs;

    // Add nodes to graph
    CHECK_CUDA(hipGraphAddKernelNode(&addNode, manual_graph, NULL, 0, &addParams));
    CHECK_CUDA(hipGraphAddKernelNode(&mulNode, manual_graph, &addNode, 1, &mulParams));
    CHECK_CUDA(hipGraphAddKernelNode(&scaleNode, manual_graph, &mulNode, 1, &scaleParams));

    // Instantiate and execute
    CHECK_CUDA(hipGraphInstantiate(&manual_graphExec, manual_graph, NULL, NULL, 0));

    CHECK_CUDA(hipEventRecord(start));

    for (int i = 0; i < iterations; i++) {
        CHECK_CUDA(hipGraphLaunch(manual_graphExec, stream));
    }

    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipStreamSynchronize(stream));
    CHECK_CUDA(hipEventSynchronize(stop));

    float ms_manual = 0;
    CHECK_CUDA(hipEventElapsedTime(&ms_manual, start, stop));

    printf("Total time: %.3f ms\n", ms_manual);
    printf("Avg per iteration: %.3f ms\n", ms_manual / iterations);
    printf("Speedup: %.2fx\n\n", ms_traditional / ms_manual);

    // ===== Method 4: Graph with Memory Operations =====
    printf("=== CUDA Graphs with Memory Operations ===\n");

    float *h_result = (float*)malloc(10 * sizeof(float));

    hipGraph_t mem_graph;
    hipGraphExec_t mem_graphExec;
    hipStream_t mem_stream;
    CHECK_CUDA(hipStreamCreate(&mem_stream));

    CHECK_CUDA(hipStreamBeginCapture(mem_stream, hipStreamCaptureModeGlobal));

    // Compute
    vectorAdd<<<gridSize, blockSize, 0, mem_stream>>>(d_a, d_b, d_c, N);

    // Copy result back (small sample)
    CHECK_CUDA(hipMemcpyAsync(h_result, d_c, 10 * sizeof(float),
                               hipMemcpyDeviceToHost, mem_stream));

    CHECK_CUDA(hipStreamEndCapture(mem_stream, &mem_graph));
    CHECK_CUDA(hipGraphInstantiate(&mem_graphExec, mem_graph, NULL, NULL, 0));

    CHECK_CUDA(hipGraphLaunch(mem_graphExec, mem_stream));
    CHECK_CUDA(hipStreamSynchronize(mem_stream));

    printf("Graph with memory operations executed successfully\n");
    printf("Sample results: %.2f, %.2f, %.2f\n\n", h_result[0], h_result[1], h_result[2]);

    // ===== Graph Update (Parameter Change) =====
    printf("=== Graph Update (Changing Parameters) ===\n");

    // Update scale parameter in existing graph
    float new_scale = 2.0f;
    void* new_scaleArgs[] = {&d_c, &new_scale, &N};
    scaleParams.kernelParams = new_scaleArgs;

    CHECK_CUDA(hipGraphExecKernelNodeSetParams(graphExec, scaleNode, &scaleParams));

    CHECK_CUDA(hipGraphLaunch(graphExec, stream));
    CHECK_CUDA(hipStreamSynchronize(stream));

    printf("Graph parameters updated successfully\n\n");

    // ===== Performance Summary =====
    printf("=== Performance Summary ===\n");
    printf("Traditional:      %.3f ms (1.00x)\n", ms_traditional);
    printf("Graph (capture):  %.3f ms (%.2fx faster)\n",
           ms_graph, ms_traditional / ms_graph);
    printf("Graph (manual):   %.3f ms (%.2fx faster)\n",
           ms_manual, ms_traditional / ms_manual);

    printf("\n=== Benefits of CUDA Graphs ===\n");
    printf("✓ Reduced CPU overhead (10μs → <1μs)\n");
    printf("✓ Better optimization opportunities\n");
    printf("✓ Predictable execution patterns\n");
    printf("✓ Ideal for inference pipelines\n");
    printf("✓ Can update parameters without reconstruction\n");

    printf("\n=== Use Cases ===\n");
    printf("• Fixed topology inference pipelines\n");
    printf("• Repeated training iterations\n");
    printf("• Real-time applications\n");
    printf("• Minimizing launch latency\n");

    // Cleanup
    CHECK_CUDA(hipGraphExecDestroy(graphExec));
    CHECK_CUDA(hipGraphExecDestroy(manual_graphExec));
    CHECK_CUDA(hipGraphExecDestroy(mem_graphExec));
    CHECK_CUDA(hipGraphDestroy(graph));
    CHECK_CUDA(hipGraphDestroy(manual_graph));
    CHECK_CUDA(hipGraphDestroy(mem_graph));
    CHECK_CUDA(hipStreamDestroy(stream));
    CHECK_CUDA(hipStreamDestroy(mem_stream));
    CHECK_CUDA(hipFree(d_a));
    CHECK_CUDA(hipFree(d_b));
    CHECK_CUDA(hipFree(d_c));
    CHECK_CUDA(hipFree(d_temp));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    free(h_result);

    printf("\nCUDA Graphs completed successfully!\n");
    return 0;
}
