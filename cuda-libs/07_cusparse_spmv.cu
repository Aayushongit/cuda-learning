// cuSPARSE SpMV: Sparse Matrix-Vector multiplication
// Essential for graph neural networks, recommendation systems, and scientific computing

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s (line %d)\n", hipGetErrorString(err), __LINE__); \
        exit(EXIT_FAILURE); \
    } \
}

#define CHECK_CUSPARSE(call) { \
    hipsparseStatus_t status = call; \
    if (status != HIPSPARSE_STATUS_SUCCESS) { \
        fprintf(stderr, "cuSPARSE Error: %d (line %d)\n", status, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
}

// Create sparse matrix in CSR format with random sparsity
void generateSparseMatrix(int rows, int cols, float density,
                         int** rowPtr, int** colIdx, float** values, int* nnz) {
    // Count non-zeros
    *nnz = 0;
    int* temp_row = (int*)malloc((rows + 1) * sizeof(int));
    temp_row[0] = 0;

    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            if ((float)rand() / RAND_MAX < density) {
                (*nnz)++;
            }
        }
        temp_row[i + 1] = *nnz;
    }

    // Allocate CSR arrays
    *rowPtr = (int*)malloc((rows + 1) * sizeof(int));
    *colIdx = (int*)malloc(*nnz * sizeof(int));
    *values = (float*)malloc(*nnz * sizeof(float));

    memcpy(*rowPtr, temp_row, (rows + 1) * sizeof(int));

    // Fill values and column indices
    int idx = 0;
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            if ((float)rand() / RAND_MAX < density) {
                (*colIdx)[idx] = j;
                (*values)[idx] = (float)(rand() % 100) / 10.0f;
                idx++;
            }
        }
    }

    free(temp_row);
}

int main() {
    const int rows = 10000;
    const int cols = 10000;
    const float density = 0.01f;  // 1% sparse (99% zeros)

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Generate sparse matrix in CSR format
    int *h_csrRowPtr, *h_csrColIdx;
    float *h_csrValues;
    int nnz;

    printf("Generating sparse matrix (%dx%d, %.1f%% density)...\n", rows, cols, density * 100);
    generateSparseMatrix(rows, cols, density, &h_csrRowPtr, &h_csrColIdx, &h_csrValues, &nnz);

    printf("Non-zero elements: %d (%.2f%% of total)\n", nnz, (float)nnz / (rows * cols) * 100);

    // Generate dense vector
    float *h_x = (float*)malloc(cols * sizeof(float));
    float *h_y = (float*)malloc(rows * sizeof(float));

    for (int i = 0; i < cols; i++) {
        h_x[i] = (float)(rand() % 100) / 10.0f;
    }

    // Device memory
    int *d_csrRowPtr, *d_csrColIdx;
    float *d_csrValues, *d_x, *d_y;

    CHECK_CUDA(hipMalloc(&d_csrRowPtr, (rows + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_csrColIdx, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_csrValues, nnz * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_x, cols * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_y, rows * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_csrRowPtr, h_csrRowPtr, (rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_csrColIdx, h_csrColIdx, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_csrValues, h_csrValues, nnz * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_x, h_x, cols * sizeof(float), hipMemcpyHostToDevice));

    // Create cuSPARSE handle
    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Create sparse matrix descriptor
    hipsparseSpMatDescr_t matA;
    CHECK_CUSPARSE(hipsparseCreateCsr(&matA, rows, cols, nnz,
                                     d_csrRowPtr, d_csrColIdx, d_csrValues,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    // Create dense vector descriptors
    hipsparseDnVecDescr_t vecX, vecY;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, cols, d_x, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, rows, d_y, HIP_R_32F));

    // Allocate buffer for SpMV
    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha, matA, vecX, &beta, vecY,
                                           HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT,
                                           &bufferSize));

    void* buffer = NULL;
    CHECK_CUDA(hipMalloc(&buffer, bufferSize));

    // Warm-up
    CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha, matA, vecX, &beta, vecY,
                               HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer));

    // Timing
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    CHECK_CUDA(hipEventRecord(start));

    // Perform SpMV: y = alpha * A * x + beta * y
    CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha, matA, vecX, &beta, vecY,
                               HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer));

    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));

    // Copy result
    CHECK_CUDA(hipMemcpy(h_y, d_y, rows * sizeof(float), hipMemcpyDeviceToHost));

    // Performance metrics
    double gflops = (2.0 * nnz * 1e-9) / (milliseconds / 1000.0);
    double bandwidth = (nnz * (sizeof(float) + sizeof(int)) + rows * sizeof(int) +
                       (rows + cols) * sizeof(float)) * 1e-9 / (milliseconds / 1000.0);

    printf("\n=== SpMV Performance ===\n");
    printf("Time: %.3f ms\n", milliseconds);
    printf("Performance: %.2f GFLOPS\n", gflops);
    printf("Bandwidth: %.2f GB/s\n", bandwidth);
    printf("Throughput: %.2f M nonzeros/sec\n", nnz / (milliseconds * 1000.0));

    // Display sample results
    printf("\nInput vector x (first 10 elements):\n");
    for (int i = 0; i < 10; i++) {
        printf("%.2f ", h_x[i]);
    }

    printf("\n\nOutput vector y (first 10 elements):\n");
    for (int i = 0; i < 10; i++) {
        printf("%.2f ", h_y[i]);
    }
    printf("\n");

    // Cleanup
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    CHECK_CUDA(hipFree(d_csrRowPtr));
    CHECK_CUDA(hipFree(d_csrColIdx));
    CHECK_CUDA(hipFree(d_csrValues));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_y));
    CHECK_CUDA(hipFree(buffer));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    free(h_csrRowPtr);
    free(h_csrColIdx);
    free(h_csrValues);
    free(h_x);
    free(h_y);

    printf("\ncuSPARSE SpMV completed successfully!\n");
    return 0;
}
