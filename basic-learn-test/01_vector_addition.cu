#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    const int n = 1024;
    size_t size = n * sizeof(float);
    
    float *h_a = (float*)malloc(size);
    float *h_b = (float*)malloc(size);
    float *h_c = (float*)malloc(size);
    
    for (int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }
    
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);
    
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    
    printf("Vector addition result (first 10 elements):\n");
    for (int i = 0; i < 10; i++) {
        printf("%.1f + %.1f = %.1f\n", h_a[i], h_b[i], h_c[i]);
    }
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    
    return 0;
}